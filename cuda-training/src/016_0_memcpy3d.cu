// #CSCS CUDA Training 
//
// #Example 16 - copy memory with hipMemcpy3D
//
// #Author Ugo Varetto
//
// #Goal: copy data between two 3D grids on the GPU
//
// #Rationale: extracting/inserting data from/to regular 3D grids is a common
//             task performed in distributed stencil computations; CUDA provides
//             facilities to ease such task through the hipMemcpy3D function
//              
//
// #Solution: invoke hipMemcpy3D specifying the various 3D parameters such as
//            offsets and extents as
//            (*row byte size*, number of rows, number of slices)
//
// #Code: 
//        1) allocate input and output grid on cpu
//        2) init cpu grid with data
//        3) allocate input and output grid on GPU and init input
//           grid by copying from cpu input grid
//        3) initialize hipMemcpy3DParms structure with details
//           of data exchange(io pointers, offsets and extent)
//        4) invoke hipMemcpy3D
//        5) copy data back to cpu and check that results are the same
//           as input grid
//
// #Compilation: nvcc -arch=sm_13 16_memcpy3d.u -o 16_memcpy_3d 
//
// #Execution: ./16_memcpy_3d
//
// #Note: the code is C++ also because the default compilation mode for CUDA is C++, all functions
//        are named with C++ convention and the syntax is checked by default against C++ grammar rules 
//
// #Note: -arch=sm_13 allows the code to run on every card with hw architecture GT200 (gtx 2xx) or better


#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "cuda_error_handler.h"

typedef double real_t;

int coord_to_idx(int x, int y, int z, int row_stride, int col_stride) {
    return x + row_stride * (y + col_stride * z);
}

void init_grid(real_t* grid, 
               int width,
               int height,
               int depth) {
    for(int k = 0; k != depth; ++k) {
        for(int j = 0; j != height; ++j) {
            for(int i = 0; i != width; ++i) {
                grid[coord_to_idx(i, j, k, width, height)] = i + j + k;
            }
        }
    }
}

#define CHECK DIE_ON_CUDA_ERROR

int main(int, char**) {

    const int width = 100; //faster if width * sizeof(real_t) = k * 512 with k a positive integer
    const int height = 200;
    const int depth = 300;
    const int size = width * height * depth;
    const int byte_size = size * sizeof(real_t);
    const int byte_row = width * sizeof(real_t);

    std::vector< real_t > h_grid_in(size, 0);
    std::vector< real_t > h_grid_out(size, 0);
    init_grid(&h_grid_in[0], width, height, depth);
    
    real_t* d_grid_in = 0;
    real_t* d_grid_out = 0;

    //Extents/positions for linear memory are ALWAYS specified as:
    // row width IN BYTES, column height IN ELEMENTS, depth IN ELEMENTS
    // fastest performance for 512 byte-aligned allocations 

    CHECK(hipMalloc(&d_grid_in, byte_size));
    CHECK(hipMemcpy(d_grid_in, &h_grid_in[0], byte_size, hipMemcpyHostToDevice));
    CHECK(hipMalloc(&d_grid_out, byte_size));
    
    hipPitchedPtr inptr = make_hipPitchedPtr(d_grid_in, byte_row, width, height);
    hipPitchedPtr outptr = make_hipPitchedPtr(d_grid_out, byte_row, width, height);

    hipMemcpy3DParms memcpyParams;
    memcpyParams.srcArray = 0;
    memcpyParams.srcPos = make_hipPos(0, 0, 0);
    memcpyParams.srcPtr = inptr;
    memcpyParams.dstArray = 0;
    memcpyParams.dstPos = make_hipPos(0, 0, 0);
    memcpyParams.dstPtr = outptr;
    memcpyParams.extent = make_hipExtent(byte_row, height, depth);
    memcpyParams.kind = hipMemcpyDeviceToDevice;

    CHECK(hipMemcpy3D(&memcpyParams));

    CHECK(hipMemcpy(&h_grid_out[0], d_grid_out, byte_size, hipMemcpyDeviceToHost));

    std::cout << std::boolalpha 
              << "Copied: " 
              << std::equal(h_grid_in.begin(), h_grid_in.end(), h_grid_out.begin())
              << std::endl;

    CHECK(hipFree(d_grid_in));
    CHECK(hipFree(d_grid_out));
    CHECK(hipDeviceReset());
    return 0;
} 