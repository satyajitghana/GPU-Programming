//Author: Ugo Varetto
//Launch multiple asynchronous memcopies on four different gpus
//and execute kernel.
//Verify (with nvvp) that transfers happen in parallel


#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <vector>
#include <algorithm>

using namespace std;

typedef signed char Int8;

__global__
void Negate(Int8* buffer) {
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    buffer[i] = -buffer[i];
}


void InitHostBuffer(Int8* buf, size_t hostSize, int numDevices) {
    const size_t devSize = hostSize / numDevices;
    assert(devSize);
    for(int i = 0; i != numDevices; ++i) {
        fill(buf + i * devSize, buf + i * devSize + devSize, Int8(-(i+1)));
    }
}


int main(int, char**) {
    assert(sizeof(Int8) == 1);
    //allocate pinned host buffer
    const size_t HOST_BUFFER_SIZE = size_t(1) << 32;
    const int NUM_DEVICES = 4;
    const size_t DEVICE_BUFFER_SIZE = HOST_BUFFER_SIZE / NUM_DEVICES;
    cout << "Number of devices:      " << NUM_DEVICES << endl
         << "Buffer size:            " << HOST_BUFFER_SIZE << endl
         << "Per-device buffer size: " << DEVICE_BUFFER_SIZE << endl;
    Int8* hostBuffer = 0;
    hipError_t err = hipHostMalloc((void**) &hostBuffer, HOST_BUFFER_SIZE, hipHostMallocDefault);
    assert(hostBuffer);
    assert(err == hipSuccess);
    //initialize host buffer with -1-1-1-1-2-2-2-2-3-3-3-3-4-4-4-4
    InitHostBuffer(hostBuffer, HOST_BUFFER_SIZE, NUM_DEVICES);
    //allocate 4 device buffers, one per device
    vector< Int8* > deviceBuffers(NUM_DEVICES, (Int8*)(0));
    for(int d = 0; d != NUM_DEVICES; ++d) {
        err = hipSetDevice(d);
        assert(err == hipSuccess);
        err = hipMalloc((void**) &deviceBuffers[d], DEVICE_BUFFER_SIZE);
        assert(deviceBuffers[d]);
        assert(err == hipSuccess);
    }
    //async per-device copies
    for(int d = 0; d != NUM_DEVICES; ++d) {
        err = hipSetDevice(d);
        assert(err == hipSuccess);
        err = hipMemcpyAsync(deviceBuffers[d], 
                              hostBuffer + d * DEVICE_BUFFER_SIZE,
                              DEVICE_BUFFER_SIZE, hipMemcpyHostToDevice);
        assert(err == hipSuccess);
    }
    //
    const int THREAD_BLOCK_SIZE = 1024;
    const int BLOCK_SIZE = DEVICE_BUFFER_SIZE / THREAD_BLOCK_SIZE;
    for(int d = 0; d != NUM_DEVICES; ++d) {
        err = hipSetDevice(d);
        assert(err == hipSuccess);
        Negate<<< BLOCK_SIZE, THREAD_BLOCK_SIZE >>>(deviceBuffers[d]);
#ifdef CHECK_KERNEL_LAUNCH       
        err == cudaGetLastError(); //no idea about what this does, does it trigger a barrier ?
        assert(err == cudaSuccess);
#endif
    }
    //
    for(int d = 0; d != NUM_DEVICES; ++d) {
        err = hipSetDevice(d);
        assert(err == hipSuccess);
        err = hipMemcpyAsync(hostBuffer + d * DEVICE_BUFFER_SIZE,
                              deviceBuffers[d], 
                              DEVICE_BUFFER_SIZE, hipMemcpyDeviceToHost);
        assert(err == hipSuccess);
    }

    for(int d = 0; d != NUM_DEVICES; ++d) {
        err = hipSetDevice(d);
        assert(err == hipSuccess);
        err = hipDeviceSynchronize();
        assert(err == hipSuccess);
    }
    
    for(int d = 0; d != NUM_DEVICES; ++d) {
        for(Int8* p = hostBuffer + d * DEVICE_BUFFER_SIZE;
            p != hostBuffer + d * DEVICE_BUFFER_SIZE + DEVICE_BUFFER_SIZE;
            ++p) assert(*p == (d + 1));
    }

    err = hipHostFree(hostBuffer);
    assert(err == hipSuccess);
    for(int d = 0; d != NUM_DEVICES; ++d) {
        err = hipSetDevice(d);
        assert(err == hipSuccess);
        err = hipFree(deviceBuffers[d]);
        assert(err == hipSuccess);
    }
    err = hipDeviceReset();
    assert(err == hipSuccess);
    cout << "PASSED" << endl;
    return 0;
}
